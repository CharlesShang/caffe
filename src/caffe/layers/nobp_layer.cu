#include "hip/hip_runtime.h"
#include <algorithm>
#include <vector>

#include "caffe/layers/nobp_layer.hpp"

namespace caffe {

template <typename Dtype>
__global__ void NoBPForward(const int n, const Dtype* in, Dtype* out) {
  CUDA_KERNEL_LOOP(index, n) {
    out[index] = in[index];
  }
}

template <typename Dtype>
void NoBPLayer<Dtype>::Forward_gpu(const vector<Blob<Dtype>*>& bottom,
    const vector<Blob<Dtype>*>& top) {
  const Dtype* bottom_data = bottom[0]->gpu_data();
  Dtype* top_data = top[0]->mutable_gpu_data();
  const int count = bottom[0]->count();
  NoBPForward<Dtype><<<CAFFE_GET_BLOCKS(count), CAFFE_CUDA_NUM_THREADS>>>(
      count, bottom_data, top_data);
  CUDA_POST_KERNEL_CHECK;
}

template <typename Dtype>
__global__ void NoBPBackward(const int n, Dtype* out_diff) {
  CUDA_KERNEL_LOOP(index, n) {
    out_diff[index] = 0;
  }
}

template <typename Dtype>
void NoBPLayer<Dtype>::Backward_gpu(const vector<Blob<Dtype>*>& top,
    const vector<bool>& propagate_down,
    const vector<Blob<Dtype>*>& bottom) {
  if (propagate_down[0]) {
    Dtype* bottom_diff = bottom[0]->mutable_gpu_diff();
    const int count = bottom[0]->count();
    // NOLINT_NEXT_LINE(whitespace/operators)
    NoBPBackward<Dtype><<<CAFFE_GET_BLOCKS(count), CAFFE_CUDA_NUM_THREADS>>>(
        count, bottom_diff);
    CUDA_POST_KERNEL_CHECK;
  }
}


INSTANTIATE_LAYER_GPU_FUNCS(NoBPLayer);


}  // namespace caffe
