#include "hip/hip_runtime.h"
#include <vector>

#include "caffe/layers/absdist_loss_layer.hpp"
#include "caffe/util/math_functions.hpp"

namespace caffe {

template <typename Dtype>
void AbsdistLossLayer<Dtype>::Forward_gpu(const vector<Blob<Dtype>*>& bottom,
    const vector<Blob<Dtype>*>& top) {
  int count = bottom[0]->count();
  caffe_gpu_sub(
      count,
      bottom[0]->gpu_data(),
      bottom[1]->gpu_data(),
      diff_.mutable_gpu_data());
  Dtype abs;
  caffe_gpu_asum(count, diff_.gpu_data(), &abs);
  Dtype loss = abs / bottom[0]->num();
  top[0]->mutable_cpu_data()[0] = loss;
  for(int i = 0; i < count; i++)
  {
    diff_.mutable_cpu_data()[i] = diff_.cpu_data()[i] > 0.0 ? 1.0 : -1.0;
  }
}

template <typename Dtype>
void AbsdistLossLayer<Dtype>::Backward_gpu(const vector<Blob<Dtype>*>& top,
    const vector<bool>& propagate_down, const vector<Blob<Dtype>*>& bottom) {
  for (int i = 0; i < 2; ++i) {
    if (propagate_down[i]) {
      const Dtype sign = (i == 0) ? 1 : -1;
      const Dtype alpha = sign * top[0]->cpu_diff()[0] / bottom[i]->num();
      caffe_gpu_axpby(
          bottom[i]->count(),              // count
          alpha,                              // alpha
          diff_.gpu_data(),                   // a
          Dtype(0),                           // beta
          bottom[i]->mutable_gpu_diff());  // b
    }
  }
}

INSTANTIATE_LAYER_GPU_FUNCS(AbsdistLossLayer);

}  // namespace caffe
